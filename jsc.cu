#include "hip/hip_runtime.h"
#include "jsc.h"

__constant__ uint4 bd[CONSTANTSIZE / sizeof(uint4)];

#define gpuerrorcheck(ans) { gpuassert((ans), __FILE__, __LINE__); }
inline void gpuassert(hipError_t code, char *file, int line, bool abort = true) {

	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void histogramproduct(dim *h1, dim *h2, dim *hr, dim hn) {

	dim tid = blockIdx.x * THREADSPERBLOCK + threadIdx.x;
	if (tid < hn) hr[tid] = h1[tid] * h2[tid];
}

__global__ void computeoutput(func f1, func f2, chunk *d1, chunk *d2, value *v1, value *v2, dim *pfxh1, dim *pfxh2, dim *pfxhp) {

	dim bx = blockIdx.x, tx = threadIdx.x;
	uint4 i = bd[bx];
	dim m = i.y ? 2 : i.z + 1;
	__shared__ dim shpfx[SHAREDSIZE / sizeof(dim)];
	chunk *shd = ((chunk *)shpfx) + CEIL(3 * m * sizeof(dim), sizeof(chunk));
	// assume THREADSPERBLOCK > m + 1
	if (tx < m && (tx || i.x)) {
		shpfx[tx] = pfxh1[i.x + tx - 1];
		shpfx[tx + m ] = pfxh2[i.x + tx - 1];
		shpfx[tx + 2 * m] = pfxhp[i.x + tx - 1];
	}
	if (!i.x) shpfx[0] = shpfx[m] = shpfx[2 * m] = 0;
	__syncthreads();

	uint3 j, l = make_uint3(shpfx[0], shpfx[m], shpfx[2 * m]);
	uint2 k;
	if (i.y) {
		j = make_uint3((shpfx[1] - l.x) / i.y, (shpfx[m + 1] - l.y) / i.y, 0);
		k = make_uint2(j.x * i.z, j.y * i.w);
	}
	else {
		j = make_uint3(shpfx[i.z] - l.x, shpfx[i.z + m] - l.y, shpfx[i.z + 2 * m] - l.z);
		k = make_uint2(0, 0);
	}

	if (i.y == i.z + 1) j.x += (shpfx[1] - l.x) % i.y;
	if (i.y == i.w + 1) j.y += (shpfx[m + 1] - l.y) % i.y;
	if (i.y) j.z = j.x * j.y;

	dim h;
	if (tx < j.x) for (h = 0; h < f1.c; h++) shd[h * j.x + tx] = d1[h * f1.n + l.x + k.x + tx];
	if (tx < j.y) for (h = 0; h < f2.c; h++) shd[j.x * f1.c + h * j.y + tx] = d2[h * f2.n + l.y + k.y + tx];

	//value *shv = (value *)(shd + j.x * f1.c + j.y * f2.c + j.z * (OUTPUTC - f1.m / BITSPERCHUNK));
	//if (tx < j.x) shv[tx] = v1[l.x + k.x + tx];
	//if (tx < j.y) shv[j.x + tx] = v2[l.y + k.y + tx];

	//if (!tx && !bx) printf("%llu %llu\n", shd[0], shd[9]);

	//if (!tx && bx == 5) printf("shd max chunks = %llu (%llu - %llu), 1 = %u (%u * %u), 2 = %u (%u * %u), %u %u\n", SHAREDSIZE / sizeof(chunk) - CEIL(3 * m * sizeof(dim), sizeof(chunk)), SHAREDSIZE / sizeof(chunk), CEIL(3 * m * sizeof(dim), sizeof(chunk)), j.x * f1.c, j.x, f1.c, j.y * f2.c, j.y, f2.c, j.z, OUTPUTC - f1.m / BITSPERCHUNK);

	__syncthreads();

	if (tx < j.z && bx == 5) {
		k.x = 0;
		for (; k.x < m - 1; k.x++)
			if (shpfx[k.x + 1 + 2 * m] - l.z > tx) break;
		k.y = tx - (shpfx[k.x + 2 * m] - l.z);
		//i = make_uint4(shpfx[k.x] - shpfx[0], shpfx[k.x + m] - shpfx[m] + j.x, shpfx[k.x + 1] - shpfx[k.x], shpfx[k.x + 1 + m] - shpfx[k.x + m]);
		i = make_uint4(shpfx[k.x], shpfx[k.x + 1], shpfx[k.x + m], shpfx[k.x + m + 1]); // fetch useful data from shared memory
		//shv[j.x + j.y + tx] = shv[] + shv[];
		i = make_uint4(i.x - l.x, i.z - l.y + j.x * f1.c, i.y - i.x, i.w - i.z);
		//printf("bx=%02u tx=%02u i.x=%02u i.y=%02u (-%02u), k.x=%02u k.y=%02u i.z=%02u i.w=%02u %02u %02u\n", bx, tx, i.x + k.y / i.w, i.y + k.y % i.w, j.x * f1.c, k.x, k.y, i.z, i.w, k.y / i.w, k.y % i.w);
		i = make_uint4(i.x + k.y / i.w, i.y + k.y % i.w, f1.m % BITSPERCHUNK, f2.s % BITSPERCHUNK);
		chunk a, b, c, t = shd[i.x + j.x * (f1.c - 1)];
		h = f2.s / BITSPERCHUNK;
		a = shd[i.y + h * j.y];
		//printf("bx=%02u tx=%02u d1=%02u d2=%02u h=%u output[%u]=%llu\n", bx, tx, i.x, i.y, h, j.x * f1.c + j.y * f2.c + h * j.z + tx, t);

		for (; h < f2.c; h++) {
			b = h == f2.c - 1 ? 0 : shd[i.y + (h + 1) * j.y];
			c = a >> i.w | b << BITSPERCHUNK - i.w;
			t = t | c << i.z;
			shd[j.x * f1.c + j.y * f2.c + h * j.z + tx] = t;
			printf("bx=%02u tx=%02u d1=%02u d2=%02u h=%u output[%u]=%llu\n", bx, tx, i.x, i.y, h, j.x * f1.c + j.y * f2.c + h * j.z + tx, t);
			t = c >> BITSPERCHUNK - i.z;
			a = b;
		}
	}
}

dim linearbinpacking(func f1, func f2, dim *hp, uint4 *o) {

	register size_t m, mb = MEMORY(0) + 3 * sizeof(dim);
	register dim a, b, c, i, j = 0, k = 0;

	for (i = 1; i <= f1.hn; i++)
		if ((m = MEMORY(i)) + mb > SHAREDSIZE || i == f1.hn) {
			a = c = CEIL(mb, SHAREDSIZE);
			do {
				b = c;
				do o[j++] = make_uint4(k, c > 1 ? c : 0, c > 1 ? c - a : i - k, c > 1 ? c - b : 0);
				while (--b);
			} while (--a);
			mb = m + 3 * sizeof(dim);
			k = i;
		}
		else mb += m;

	return j;
}

int main(int argc, char *argv[]) {

	func f1, f2;
	struct timeval t1, t2;
	init_genrand64(SEED);
	srand(SEED);

	f1.n = 100;
	f1.m = 80;
	f2.n = 30;
	f2.m = 100;

	f1.c = CEIL(f1.m, BITSPERCHUNK);
	f2.c = CEIL(f2.m, BITSPERCHUNK);
	f1.vars = (var *)malloc(sizeof(var) * f1.m);
	f2.vars = (var *)malloc(sizeof(var) * f2.m);
        f1.v = (value *)malloc(sizeof(value) * f1.n);
        f2.v = (value *)malloc(sizeof(value) * f2.n);
	f1.data = (chunk *)calloc(1, sizeof(chunk) * f1.n * f1.c);
	f2.data = (chunk *)calloc(1, sizeof(chunk) * f2.n * f2.c);

	if (!f1.data || !f2.data) {
		printf("Not enough memory!\n");
		return 1;
	}

	printf("Random data... ");
	fflush(stdout);
	gettimeofday(&t1, NULL);
        randomvars(f1);
        randomvars(f2);
        randomdata(f1);
        randomdata(f2);
        randomvalues(f1);
        randomvalues(f2);
	gettimeofday(&t2, NULL);
	printf("%f seconds\n", (double)(t2.tv_usec - t1.tv_usec) / 1e6 + t2.tv_sec - t1.tv_sec);

	chunk *c1 = (chunk *)calloc(f1.c, sizeof(chunk));
	chunk *c2 = (chunk *)calloc(f2.c, sizeof(chunk));
	sharedmasks(&f1, c1, &f2, c2);

	f1.mask = f2.mask = (1ULL << (f1.s % BITSPERCHUNK)) - 1;
	printf("%u shared variables\n", f1.s);
	if (!f1.s) return 1;

	printf("Shift & Reorder... ");
	fflush(stdout);
	gettimeofday(&t1, NULL);
	shared2least(f1, c1);
	shared2least(f2, c2);
	reordershared(f2, f1.vars);
	gettimeofday(&t2, NULL);
	printf("%f seconds\n", (double)(t2.tv_usec - t1.tv_usec) / 1e6 + t2.tv_sec - t1.tv_sec);

	printf("Sort... ");
	fflush(stdout);
	gettimeofday(&t1, NULL);
	sort(f1);
	sort(f2);
	gettimeofday(&t2, NULL);
	printf("%f seconds\n", (double)(t2.tv_usec - t1.tv_usec) / 1e6 + t2.tv_sec - t1.tv_sec);

	print(f1, c1);
	print(f2, c2);

	printf("%u unique combinations\n", f1.hn = uniquecombinations(f1));
	printf("%u unique combinations\n", f2.hn = uniquecombinations(f2));
	f1.h = (dim *)calloc(f1.hn, sizeof(dim));
	f2.h = (dim *)calloc(f2.hn, sizeof(dim));

	printf("Histogram... ");
	fflush(stdout);
	gettimeofday(&t1, NULL);
	histogram(f1);
	histogram(f2);
	gettimeofday(&t2, NULL);
	printf("%f seconds\n", (double)(t2.tv_usec - t1.tv_usec) / 1e6 + t2.tv_sec - t1.tv_sec);

	printf("Matching Rows... ");
	fflush(stdout);
	gettimeofday(&t1, NULL);
	f1.hmask = (chunk *)calloc(CEIL(f1.hn, BITSPERCHUNK), sizeof(chunk));
	f2.hmask = (chunk *)calloc(CEIL(f2.hn, BITSPERCHUNK), sizeof(chunk));
	dim n1, n2, hn;
	markmatchingrows(f1, f2, &n1, &n2, &hn);
	copymatchingrows(&f1, &f2, n1, n2, hn);
	gettimeofday(&t2, NULL);
	printf("%f seconds\n", (double)(t2.tv_usec - t1.tv_usec) / 1e6 + t2.tv_sec - t1.tv_sec);

	printf("%u matching rows\n", f1.n);
	printf("%u matching rows\n", f2.n);

	chunk *d1d, *d2d;
	value *v1d, *v2d;
	dim on, *h1d, *h2d, *hpd, *pfxh1d, *pfxh2d, *pfxhpd;
	printf("Allocating... ");
	fflush(stdout);
	gettimeofday(&t1, NULL);
	hipMalloc(&d1d, sizeof(chunk) * f1.n * f1.c);
	hipMalloc(&d2d, sizeof(chunk) * f2.n * f2.c);
	hipMalloc(&v1d, sizeof(value) * f1.n);
        hipMalloc(&v2d, sizeof(value) * f2.n);
	hipMalloc(&h1d, sizeof(dim) * hn);
	hipMalloc(&h2d, sizeof(dim) * hn);
	hipMalloc(&hpd, sizeof(dim) * hn);
        hipMalloc(&pfxh1d, sizeof(dim) * hn);
        hipMalloc(&pfxh2d, sizeof(dim) * hn);
        hipMalloc(&pfxhpd, sizeof(dim) * hn);
	gettimeofday(&t2, NULL);
        printf("%f seconds\n", (double)(t2.tv_usec - t1.tv_usec) / 1e6 + t2.tv_sec - t1.tv_sec);

	hipMemcpy(d1d, f1.data, sizeof(chunk) * f1.n * f1.c, hipMemcpyHostToDevice);
	hipMemcpy(d2d, f2.data, sizeof(chunk) * f2.n * f2.c, hipMemcpyHostToDevice);
        hipMemcpy(v1d, f1.v, sizeof(value) * f1.n, hipMemcpyHostToDevice);
        hipMemcpy(v2d, f2.v, sizeof(value) * f2.n, hipMemcpyHostToDevice);
	hipMemcpy(h1d, f1.h, sizeof(dim) * hn, hipMemcpyHostToDevice);
	hipMemcpy(h2d, f2.h, sizeof(dim) * hn, hipMemcpyHostToDevice);

	histogramproduct<<<CEIL(hn, THREADSPERBLOCK), THREADSPERBLOCK>>>(h1d, h2d, hpd, hn);

	CUDPPHandle cudpp, pfxsum = 0;
	cudppCreate(&cudpp);
	CUDPPConfiguration config;
	config.op = CUDPP_ADD;
	config.datatype = CUDPP_UINT;
	config.algorithm = CUDPP_SCAN;
	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
	cudppPlan(cudpp, &pfxsum, config, hn, 1, 0);
	cudppScan(pfxsum, pfxh1d, h1d, hn);
	cudppScan(pfxsum, pfxh2d, h2d, hn);
	cudppScan(pfxsum, pfxhpd, hpd, hn);
	cudppDestroyPlan(pfxsum);
	cudppDestroy(cudpp);

	hipMemcpy(&on, pfxhpd + hn - 1, sizeof(dim), hipMemcpyDeviceToHost);
	printf("Result size = %zu bytes (%u lines)\n", sizeof(chunk) * on * OUTPUTC, on);

	dim hp[hn], bn;
	uint4 *bh = (uint4 *)malloc(sizeof(uint4) * on);
	hipMemcpy(hp, hpd, sizeof(dim) * hn, hipMemcpyDeviceToHost);
	bn = linearbinpacking(f1, f2, hp, bh);
	bh = (uint4 *)realloc(bh, sizeof(uint4) * bn);
	hipMemcpyToSymbol(HIP_SYMBOL(bd), bh, sizeof(uint4) * bn);
	printf("Used constant memory = %zu bytes\n", sizeof(uint3) * bn);

	dim i;
	for (i = 0; i < hn; i++) printf("%u * %u = %u (%zu)\n", f1.h[i], f2.h[i], hp[i], MEMORY(i));
	for (i = 0; i < bn; i++) printf("%u %u %u %u\n", bh[i].x, bh[i].y, bh[i].z, bh[i].w);

	computeoutput<<<bn, THREADSPERBLOCK>>>(f1, f2, d1d, d2d, v1d, v2d, pfxh1d, pfxh2d, pfxhpd);
	gpuerrorcheck(hipPeekAtLastError());
	gpuerrorcheck(hipDeviceSynchronize());

	puts("Checksum...");
	printf("Checksum 1 = %u (size = %zu bytes)\n", crc32(f1.data, sizeof(chunk) * f1.n * f1.c), sizeof(chunk) * f1.n * f1.c);
	printf("Checksum Histogram 1 = %u (size = %zu bytes)\n", crc32(f1.h, sizeof(dim) * f1.hn), sizeof(dim) * f1.hn);
	printf("Checksum 2 = %u (size = %zu bytes)\n", crc32(f2.data, sizeof(chunk) * f2.n * f2.c), sizeof(chunk) * f2.n * f2.c);
	printf("Checksum Histogram 2 = %u (size = %zu bytes)\n", crc32(f2.h, sizeof(dim) * f2.hn), sizeof(dim) * f2.hn);

	hipFree(d1d);
	hipFree(d2d);
	hipFree(v1d);
	hipFree(v2d);
	hipFree(h1d);
	hipFree(h2d);
	hipFree(hpd);
	hipFree(pfxh1d);
	hipFree(pfxh2d);
        hipFree(pfxhpd);

	free(f1.hmask);
	free(f2.hmask);
	free(f1.vars);
	free(f1.data);
	free(f2.vars);
	free(f2.data);
	free(f1.h);
	free(f2.h);
        free(f1.v);
        free(f2.v);

	return 0;
}
