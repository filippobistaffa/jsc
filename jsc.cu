#include "hip/hip_runtime.h"
#include "jsc.h"

#ifdef PRINTTIME
static struct timeval t1, t2;
#endif

__constant__ uint3 bd[CONSTANTSIZE / sizeof(uint3)];

__global__ void histogramproductkernel(dim *h1, dim *h2, dim *hr, dim hn) {

	dim tid = blockIdx.x * THREADSPERBLOCK + threadIdx.x;
	if (tid < hn) hr[tid] = h1[tid] * h2[tid];
}

__global__ void jointsumkernel(func f1, func f2, func f3, chunk *d1, chunk *d2, chunk *d3, value *v1, value *v2, value *v3, dim *pfxh1, dim *pfxh2, dim *pfxhp) {

	dim bx = blockIdx.x, tx = threadIdx.x;
	uint2 k;
	uint3 j, l, o = bd[bx];
	uint4 i = make_uint4(o.x, o.y, o.z / max(o.y, 1), o.z % max(o.y, 1));
	dim h, m = i.y ? 2 : i.z + 1;
	__shared__ dim shpfx[SHAREDSIZE / sizeof(dim)];
	chunk *shd = ((chunk *)shpfx) + CEIL(3 * m * sizeof(dim), sizeof(chunk));

	assert(THREADSPERBLOCK >= m);
	if (tx < m && (tx || i.x)) {
		shpfx[tx] = pfxh1[i.x + tx - 1];
		shpfx[tx + m ] = pfxh2[i.x + tx - 1];
		shpfx[tx + 2 * m] = pfxhp[i.x + tx - 1];
	}
	if (!i.x) shpfx[0] = shpfx[m] = shpfx[2 * m] = 0;
	__syncthreads();

	l = make_uint3(shpfx[0], shpfx[m], shpfx[2 * m]);
	o.x = 0;

	if (i.y) {
		j = make_uint3((o.y = shpfx[1] - l.x) / i.y, (o.z = shpfx[m + 1] - l.y) / i.y, 0);
		k = make_uint2(j.x * i.z, j.y * i.w);
	}
	else {
		j = make_uint3(shpfx[i.z] - l.x, shpfx[i.z + m] - l.y, shpfx[i.z + 2 * m] - l.z);
		k = make_uint2(0, 0);
	}

	if (i.y) o.x = (i.y * i.z + i.w) * j.x * j.y + i.z * j.x * (o.z % i.y);
	if (i.y == i.z + 1) {
		j.x += o.y % i.y;
		o.x += i.w * j.y * (o.y % i.y);
	}
	if (i.y == i.w + 1) j.y += o.z % i.y;
	if (i.y) j.z = j.x * j.y;
	assert(THREADSPERBLOCK >= j.z);

	// j.x = number of lines for input 1
	// j.y = number of lines for input 2
	// j.z = number of lines for output

	//if (tx < j.x * f1.c) shd[tx] = d1[(tx / j.x) * f1.n + l.x + k.x + tx % j.x];
	//if (tx < j.y * f2.c) shd[j.x * f1.c + tx] = d2[(tx / j.y) * f2.n + l.y + k.y + tx % j.y];
	if (tx < j.x) for (h = 0; h < f1.c; h++) shd[h * j.x + tx] = d1[h * f1.n + l.x + k.x + tx];
	if (tx < j.y) for (h = 0; h < f2.c; h++) shd[j.x * f1.c + h * j.y + tx] = d2[h * f2.n + l.y + k.y + tx];

	value *shv = (value *)(shd + j.x * f1.c + j.y * f2.c + j.z * (f3.c - f1.m / BITSPERCHUNK));
	if (tx < j.x) { shv[tx] = v1[l.x + k.x + tx]; /*printf("[%02u] (1) shv[%02u] <- %f\n", tx, tx, v1[l.x + k.x + tx]);*/ }
	if (tx < j.y) { shv[j.x + tx] = v2[l.y + k.y + tx]; /*printf("[%02u] (2) shv[%02u] <- %f\n", tx, j.x + tx, v2[l.y + k.y + tx]);*/ }

	__syncthreads();

	if (tx < j.z) {
		o.y = 0;
		if (i.y || i.z == 1) {
			i = make_uint4(0, j.x * f1.c, j.x, j.y);
			o.z = tx;
			h = j.x;
		} else {
			for (; o.y < m - 1; o.y++) if (shpfx[o.y + 1 + 2 * m] - l.z > tx) break;
			o.z = tx - (shpfx[o.y + 2 * m] - l.z);
			i = make_uint4(shpfx[o.y], shpfx[o.y + 1], shpfx[o.y + m], shpfx[o.y + m + 1]); // fetch useful data from shared memory
			h = i.z - l.y + j.x;
			i = make_uint4(i.x - l.x, i.z - l.y + j.x * f1.c, i.y - i.x, i.w - i.z);
		}
		// o.y = which of the n groups of this block this thread belongs
		// o.z = index of this thread w.r.t. his group (in this block)
		// i.x = start of input 1 row for this group
		// i.y = start of input 2 row for this group
		// i.z = total number of input 1 rows for this group
		// i.w = total number of input 2 rows for this group

		//shv[j.x + j.y + tx] = shv[i.x + o.z / i.w] + shv[h + o.z % i.w];
		JOINTOPERATION(shv[j.x + j.y + tx], shv[i.x + o.z / i.w], shv[h + o.z % i.w]);
		//printf("[%02u] shv[%u] = shv[%u] + shv[%u] = %f = %f + %f\n", tx, j.x + j.y + tx, i.x + o.z / i.w, h + o.z % i.w, shv[j.x + j.y + tx], shv[i.x + o.z / i.w], shv[h + o.z % i.w]);
		i = make_uint4(i.x + o.z / i.w, i.y + o.z % i.w, f1.m % BITSPERCHUNK, f2.s % BITSPERCHUNK);
		chunk a, b, c, t = i.z ? shd[i.x + j.x * (f1.c - 1)] : 0;
		//printf("[%02u] t=%lu\n",tx, t);
		h = f2.s / BITSPERCHUNK;
		a = shd[i.y + h * j.y];
		for (; h < f2.c; h++) {
			b = h == f2.c - 1 ? 0 : shd[i.y + (h + 1) * j.y];
			c = a >> i.w | b << BITSPERCHUNK - i.w;
			t = t | c << i.z;
			shd[j.x * f1.c + j.y * f2.c + (h - f1.m / BITSPERCHUNK) * j.z + tx] = t;
			//printf("bx=%02u tx=%02u d1=%02u d2=%02u (-%02u) h=%u output[%u]=%llu\n", bx, tx, i.x, i.y, j.x *f1.c, h, j.x * f1.c + j.y * f2.c + h * j.z + tx, t);
			t = c >> BITSPERCHUNK - i.z;
			a = b;
		}

		v3[l.z + o.x + tx] = shv[j.x + j.y + tx];
		//printf("[%02u] v[%u] = shv[%u] = %f\n", tx, l.z + o.x + tx, j.x + j.y + tx, v3[l.z + o.x + tx]);
		for (h = 0; h < f1.m / BITSPERCHUNK; h++) {
			d3[l.z + o.x + h * f3.n + tx] = shd[i.x + h * j.x];
			//printf("[%u] (1) %u <- (%u)\n", tx, l.z + o.x + h * f3.n + tx, i.x + h * j.x);
		}
		for (; h < f3.c; h++) {
			d3[l.z + o.x + h * f3.n + tx] = shd[j.x * f1.c + j.y * f2.c + (h - f1.m / BITSPERCHUNK) * j.z + tx];
			//printf("[%u] (2) %u <- (%u)\n", tx, l.z + o.x + h * f3.n + tx, j.x * f1.c + j.y * f2.c + (h - f1.m / BITSPERCHUNK) * j.z + tx);
		}
	}
}

dim linearbinpacking(func *f1, func *f2, dim *hp, uint3 *o) {

	register dim b, c, i, t, j = 0, k = 0, tb = hp[0];
	register size_t m, mb = MEMORY(0) + 3 * sizeof(dim);

	for (i = 1; i <= f1->hn; i++)
		if ((m = MEMORY(i)) + mb > SHAREDSIZE | (t = hp[i]) + tb > THREADSPERBLOCK || i == f1->hn) {
			c = (m + mb > SHAREDSIZE) ? CEIL(mb, SHAREDSIZE) : CEIL(tb, THREADSPERBLOCK);
			b = c * c;
			do o[j++] = make_uint3(k, c > 1 ? c : 0, c > 1 ? c * c - b : i - k);
			while (--b);
			mb = m + 3 * sizeof(dim);
			tb = t;
			k = i;
		}
		else mb += m, tb += t;

	return j;
}

func jointsum(func *f1, func *f2) {

	#ifdef FUNCTIONCODE
	register id i;

	printf("f1.n = %u;\nf1.m = %u;\n", f1->n, f1->m);
	printf("chunk data1[] = {%lu", f1->data[0]);
	for (i = 1; i < f1->c * f1->n; i++)
		printf(",%lu", f1->data[i]);
	puts("};");
	printf("value v1[] = {%f", f1->v[0]);
	for (i = 1; i < f1->n; i++)
		printf(",%f", f1->v[i]);
	puts("};");
	printf("id vars1[] = {%u", f1->vars[0]);
	for (i = 1; i < f1->m; i++)
		printf(",%u", f1->vars[i]);
	puts("};");

	printf("f2.n = %u;\nf2.m = %u;\n", f2->n, f2->m);
	printf("chunk data2[] = {%lu", f2->data[0]);
        for (i = 1; i < f2->c * f2->n; i++)
                printf(",%lu", f2->data[i]);
        puts("};");
        printf("value v2[] = {%f", f2->v[0]);
        for (i = 1; i < f2->n; i++)
                printf(",%f", f2->v[i]);
        puts("};");
        printf("id vars2[] = {%u", f2->vars[0]);
        for (i = 1; i < f2->m; i++)
                printf(",%u", f2->vars[i]);
        puts("};");
	#endif

	register func f3;
	register chunk *c1 = (chunk *)calloc(f1->c, sizeof(chunk));
	register chunk *c2 = (chunk *)calloc(f2->c, sizeof(chunk));
	sharedmasks(f1, c1, f2, c2);

	f1->mask = f2->mask = f3.mask = (1ULL << (f1->s % BITSPERCHUNK)) - 1;
	#ifdef PRINTINFO
	printf("%u shared variables\n", f1->s);
	#endif
	//if (!f1->s) return 1;
	f3.s = f1->s;

	TIMER_START("Shift & Reorder...");
	shared2least(*f1, c1);
	shared2least(*f2, c2);
	reordershared(*f2, f1->vars);
	TIMER_STOP;

	TIMER_START("Sort...");
	sort(*f1);
	sort(*f2);
	TIMER_STOP;

	f1->hn = uniquecombinations(*f1);
	f2->hn = uniquecombinations(*f2);
	#ifdef PRINTINFO
	printf("%u unique combinations\n", f1->hn);
	printf("%u unique combinations\n", f2->hn);
	#endif
	f1->h = (dim *)calloc(f1->hn, sizeof(dim));
	f2->h = (dim *)calloc(f2->hn, sizeof(dim));

	TIMER_START("Histogram...");
	histogram(*f1);
	histogram(*f2);
	TIMER_STOP;

	TIMER_START("Matching Rows...");
	f1->hmask = (chunk *)calloc(CEIL(f1->hn, BITSPERCHUNK), sizeof(chunk));
	f2->hmask = (chunk *)calloc(CEIL(f2->hn, BITSPERCHUNK), sizeof(chunk));
	dim n1, n2, hn;
	markmatchingrows(*f1, *f2, &n1, &n2, &hn);
	copymatchingrows(f1, f2, n1, n2, hn);
	TIMER_STOP;

	#ifdef PRINTINFO
	printf("%u matching rows\n", f1->n);
	print(*f1);
	printf("%u matching rows\n", f2->n);
	print(*f2);
	#endif

	assert(f1->n && f2->n);

	chunk *d1d, *d2d, *d3d;
	value *v1d, *v2d, *v3d;
	dim *h1d, *h2d, *hpd, *pfxh1d, *pfxh2d, *pfxhpd;
	TIMER_START("Allocating... ");
	hipMalloc(&d1d, sizeof(chunk) * f1->n * f1->c);
	hipMalloc(&d2d, sizeof(chunk) * f2->n * f2->c);
	hipMalloc(&v1d, sizeof(value) * f1->n);
        hipMalloc(&v2d, sizeof(value) * f2->n);
	hipMalloc(&h1d, sizeof(dim) * hn);
	hipMalloc(&h2d, sizeof(dim) * hn);
	hipMalloc(&hpd, sizeof(dim) * hn);
        hipMalloc(&pfxh1d, sizeof(dim) * hn);
        hipMalloc(&pfxh2d, sizeof(dim) * hn);
        hipMalloc(&pfxhpd, sizeof(dim) * hn);
	TIMER_STOP;

	hipMemcpy(d1d, f1->data, sizeof(chunk) * f1->n * f1->c, hipMemcpyHostToDevice);
	hipMemcpy(d2d, f2->data, sizeof(chunk) * f2->n * f2->c, hipMemcpyHostToDevice);
        hipMemcpy(v1d, f1->v, sizeof(value) * f1->n, hipMemcpyHostToDevice);
        hipMemcpy(v2d, f2->v, sizeof(value) * f2->n, hipMemcpyHostToDevice);
	hipMemcpy(h1d, f1->h, sizeof(dim) * hn, hipMemcpyHostToDevice);
	hipMemcpy(h2d, f2->h, sizeof(dim) * hn, hipMemcpyHostToDevice);

	histogramproductkernel<<<CEIL(hn, THREADSPERBLOCK), THREADSPERBLOCK>>>(h1d, h2d, hpd, hn);
	gpuerrorcheck(hipPeekAtLastError());
	gpuerrorcheck(hipDeviceSynchronize());

	// Determine temporary device storage requirements for inclusive prefix sum
	void *ts = NULL;
	size_t tsn = 0;

	hipcub::DeviceScan::InclusiveSum(ts, tsn, h1d, pfxh1d, hn);
	#ifdef PRINTSIZE
	printf("Temporary storage for prefix sum = %zu bytes\n", tsn);
	#endif
	hipMalloc(&ts, tsn);
	hipcub::DeviceScan::InclusiveSum(ts, tsn, h1d, pfxh1d, hn);
	hipFree(ts);

	ts = NULL;
	tsn = 0;
	hipcub::DeviceScan::InclusiveSum(ts, tsn, h2d, pfxh2d, hn);
	#ifdef PRINTSIZE
	printf("Temporary storage for prefix sum = %zu bytes\n", tsn);
	#endif
	hipMalloc(&ts, tsn);
	hipcub::DeviceScan::InclusiveSum(ts, tsn, h2d, pfxh2d, hn);
	hipFree(ts);

	ts = NULL;
	tsn = 0;
	hipcub::DeviceScan::InclusiveSum(ts, tsn, hpd, pfxhpd, hn);
	hipMalloc(&ts, tsn);
	hipcub::DeviceScan::InclusiveSum(ts, tsn, hpd, pfxhpd, hn);
	hipFree(ts);

	hipMemcpy(&f3.n, pfxhpd + hn - 1, sizeof(dim), hipMemcpyDeviceToHost);
	f3.m = f1->m + f2->m - f1->s;

	ALLOCFUNC(f3, chunk, id, value);
	#ifdef PRINTSIZE
	printf("Result size = %zu bytes (%u lines)\n", sizeof(chunk) * f3.n * f3.c, f3.n);
	#endif
	hipMalloc(&d3d, sizeof(chunk) * f3.n * f3.c);
	hipMalloc(&v3d, sizeof(value) * f3.n);
	memcpy(f3.vars, f1->vars, sizeof(id) * f1->m);
	memcpy(f3.vars + f1->m, f2->vars + f2->s, sizeof(id) * (f2->m - f1->s));

	dim hp[hn], bn;
	uint3 *bh = (uint3 *)malloc(sizeof(uint3) * f3.n);
	hipMemcpy(hp, hpd, sizeof(dim) * hn, hipMemcpyDeviceToHost);

	// bn = number of blocks needed
	// each bh[i] stores the information regarding the "i"-th block
	// .x =
	// .y =
	// .z =

	bn = linearbinpacking(f1, f2, hp, bh);
	bh = (uint3 *)realloc(bh, sizeof(uint3) * bn);
	#ifdef PRINTSIZE
	printf("%u blocks needed\n", bn);
	printf("Needed constant memory = %zu bytes (Max = %u bytes)\n", sizeof(uint3) * bn, CONSTANTSIZE);
	#endif
	assert(CONSTANTSIZE > sizeof(uint3) * bn);
	hipMemcpyToSymbol(HIP_SYMBOL(bd), bh, sizeof(uint3) * bn);

	//dim i;
	//for (i = 0; i < hn; i++) printf("%u * %u = %u (%zu bytes)\n", f1->h[i], f2->h[i], hp[i], MEMORY(i));
	//for (i = 0; i < bn; i++) printf("%u %u %u\n", bh[i].x, bh[i].y, bh[i].z);

	jointsumkernel<<<bn, THREADSPERBLOCK>>>(*f1, *f2, f3, d1d, d2d, d3d, v1d, v2d, v3d, pfxh1d, pfxh2d, pfxhpd);
	gpuerrorcheck(hipPeekAtLastError());
	gpuerrorcheck(hipDeviceSynchronize());

	hipMemcpy(f3.data, d3d, sizeof(chunk) * f3.n * f3.c, hipMemcpyDeviceToHost);
	hipMemcpy(f3.v, v3d, sizeof(value) * f3.n, hipMemcpyDeviceToHost);

	// Order output table for debugging purposes
	//f3.s = f3.m;
	//f3.mask = (1ULL << (f3.s % BITSPERCHUNK)) - 1;
	//sort(f3);
	//print(f1, NULL);
	//print(f2, NULL);
	//print(f3, NULL);

	#ifdef PRINTCHECKSUM
	puts("Checksum...");
	printf("Checksum Data 1 = %u (size = %zu bytes)\n", crc32(f1->data, sizeof(chunk) * f1->n * f1->c), sizeof(chunk) * f1->n * f1->c);
	printf("Checksum Values 1 = %u (size = %zu bytes)\n", crc32(f1->v, sizeof(value) * f1->n), sizeof(value) * f1->n);
	printf("Checksum Histogram 1 = %u (size = %zu bytes)\n", crc32(f1->h, sizeof(dim) * f1->hn), sizeof(dim) * f1->hn);
	printf("Checksum Data 2 = %u (size = %zu bytes)\n", crc32(f2->data, sizeof(chunk) * f2->n * f2->c), sizeof(chunk) * f2->n * f2->c);
	printf("Checksum Values 2 = %u (size = %zu bytes)\n", crc32(f2->v, sizeof(value) * f2->n), sizeof(value) * f2->n);
	printf("Checksum Histogram 2 = %u (size = %zu bytes)\n", crc32(f2->h, sizeof(dim) * f2->hn), sizeof(dim) * f2->hn);
	printf("Checksum Output Data = %u (size = %zu bytes)\n", crc32(f3.data, sizeof(chunk) * f3.n * f3.c), sizeof(chunk) * f3.n * f3.c);
	printf("Checksum Output Values = %u (size = %zu bytes)\n", crc32(f3.v, sizeof(value) * f3.n), sizeof(value) * f3.n);
	#endif

	hipFree(d1d);
	hipFree(d2d);
	hipFree(d3d);
	hipFree(v1d);
	hipFree(v2d);
	hipFree(v3d);
	hipFree(h1d);
	hipFree(h2d);
	hipFree(hpd);
	hipFree(pfxh1d);
	hipFree(pfxh2d);
        hipFree(pfxhpd);
	free(f1->hmask);
	free(f2->hmask);
	free(f1->h);
	free(f2->h);
	free(c1);
	free(c2);
	free(bh);

	return f3;
}

#ifdef JSCMAIN

int main(int argc, char *argv[]) {

	func f1, f2, f3;
	init_genrand64(SEED);
	srand(SEED);

	f1.n = 1000;
	f1.m = 80;
	f2.n = 3000;
	f2.m = 100;
	ALLOCFUNC(f1, chunk, id, value);
	ALLOCFUNC(f2, chunk, id, value);

	if (!f1.data || !f2.data) {
		printf("Not enough memory!\n");
		return 1;
	}

	printf("Random data... ");
	fflush(stdout);
	gettimeofday(&t1, NULL);
        randomvars(f1);
        randomvars(f2);
        randomdata(f1);
        randomdata(f2);
        randomvalues(f1);
        randomvalues(f2);
	gettimeofday(&t2, NULL);
	printf("%f seconds\n", (double)(t2.tv_usec - t1.tv_usec) / 1e6 + t2.tv_sec - t1.tv_sec);

	f3 = jointsum(&f1, &f2);

	FREEFUNC(f1);
	FREEFUNC(f2);
	FREEFUNC(f3);

	return 0;
}

#endif
